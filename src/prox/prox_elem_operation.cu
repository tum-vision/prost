/**
* This file is part of prost.
*
* Copyright 2016 Thomas Möllenhoff <thomas dot moellenhoff at in dot tum dot de> 
* and Emanuel Laude <emanuel dot laude at in dot tum dot de> (Technical University of Munich)
*
* prost is free software: you can redistribute it and/or modify
* it under the terms of the GNU General Public License as published by
* the Free Software Foundation, either version 3 of the License, or
* (at your option) any later version.
*
* prost is distributed in the hope that it will be useful,
* but WITHOUT ANY WARRANTY; without even the implied warranty of
* MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
* GNU General Public License for more details.
*
* You should have received a copy of the GNU General Public License
* along with prost. If not, see <http://www.gnu.org/licenses/>.
*/

#include "prost/prox/prox_elem_operation.hpp"
#include "prost/prox/prox_elem_operation.inl"

#include "prost/prox/elemop/elem_operation.hpp"
#include "prost/prox/elemop/elem_operation_1d.hpp"
#include "prost/prox/elemop/elem_operation_norm2.hpp"
#include "prost/prox/elemop/elem_operation_ind_simplex.hpp"
#include "prost/prox/elemop/elem_operation_ind_sum.hpp"
#include "prost/prox/elemop/elem_operation_singular_nx2.hpp"
#include "prost/prox/elemop/elem_operation_eigen_2x2.hpp"
#include "prost/prox/elemop/elem_operation_eigen_3x3.hpp"
#include "prost/prox/elemop/elem_operation_mass_norm.hpp"
#include "prost/prox/elemop/function_1d.hpp"
#include "prost/prox/elemop/function_2d.hpp"

namespace prost {
  
// Explicit template instantiation
// TODO: simplify this

// float
// ElemOperation1D
template class ProxElemOperation<float, ElemOperation1D<float, Function1DZero<float>>>;
template class ProxElemOperation<float, ElemOperation1D<float, Function1DAbs<float>>>;
template class ProxElemOperation<float, ElemOperation1D<float, Function1DSquare<float>>>;
template class ProxElemOperation<float, ElemOperation1D<float, Function1DIndLeq0<float>>>;
template class ProxElemOperation<float, ElemOperation1D<float, Function1DIndGeq0<float>>>;
template class ProxElemOperation<float, ElemOperation1D<float, Function1DIndEq0<float>>>;
template class ProxElemOperation<float, ElemOperation1D<float, Function1DIndBox01<float>>>;
template class ProxElemOperation<float, ElemOperation1D<float, Function1DMaxPos0<float>>>;
template class ProxElemOperation<float, ElemOperation1D<float, Function1DL0<float>>>;
template class ProxElemOperation<float, ElemOperation1D<float, Function1DHuber<float>>>;
template class ProxElemOperation<float, ElemOperation1D<float, Function1DLq<float>>>;
template class ProxElemOperation<float, ElemOperation1D<float, Function1DLqPlusEps<float>>>;
template class ProxElemOperation<float, ElemOperation1D<float, Function1DTruncQuad<float>>>;
template class ProxElemOperation<float, ElemOperation1D<float, Function1DTruncLinear<float>>>;

// ElemOperationNorm2
template class ProxElemOperation<float, ElemOperationNorm2<float, Function1DZero<float>>>;
template class ProxElemOperation<float, ElemOperationNorm2<float, Function1DAbs<float>>>;
template class ProxElemOperation<float, ElemOperationNorm2<float, Function1DSquare<float>>>;
template class ProxElemOperation<float, ElemOperationNorm2<float, Function1DIndLeq0<float>>>;
template class ProxElemOperation<float, ElemOperationNorm2<float, Function1DIndGeq0<float>>>;
template class ProxElemOperation<float, ElemOperationNorm2<float, Function1DIndEq0<float>>>;
template class ProxElemOperation<float, ElemOperationNorm2<float, Function1DIndBox01<float>>>;
template class ProxElemOperation<float, ElemOperationNorm2<float, Function1DMaxPos0<float>>>;
template class ProxElemOperation<float, ElemOperationNorm2<float, Function1DL0<float>>>;
template class ProxElemOperation<float, ElemOperationNorm2<float, Function1DHuber<float>>>;
template class ProxElemOperation<float, ElemOperationNorm2<float, Function1DLq<float>>>;
template class ProxElemOperation<float, ElemOperationNorm2<float, Function1DLqPlusEps<float>>>;
template class ProxElemOperation<float, ElemOperationNorm2<float, Function1DTruncQuad<float>>>;
template class ProxElemOperation<float, ElemOperationNorm2<float, Function1DTruncLinear<float>>>;


// ElemOperationSingularNx2<Function2DSum1D>
template class ProxElemOperation<float, ElemOperationSingularNx2<float, Function2DSum1D<float, Function1DZero<float>>>>;
template class ProxElemOperation<float, ElemOperationSingularNx2<float, Function2DSum1D<float, Function1DAbs<float>>>>;
template class ProxElemOperation<float, ElemOperationSingularNx2<float, Function2DSum1D<float, Function1DSquare<float>>>>;
template class ProxElemOperation<float, ElemOperationSingularNx2<float, Function2DSum1D<float, Function1DIndLeq0<float>>>>;
template class ProxElemOperation<float, ElemOperationSingularNx2<float, Function2DSum1D<float, Function1DIndGeq0<float>>>>;
template class ProxElemOperation<float, ElemOperationSingularNx2<float, Function2DSum1D<float, Function1DIndEq0<float>>>>;
template class ProxElemOperation<float, ElemOperationSingularNx2<float, Function2DSum1D<float, Function1DIndBox01<float>>>>;
template class ProxElemOperation<float, ElemOperationSingularNx2<float, Function2DSum1D<float, Function1DMaxPos0<float>>>>;
template class ProxElemOperation<float, ElemOperationSingularNx2<float, Function2DSum1D<float, Function1DL0<float>>>>;
template class ProxElemOperation<float, ElemOperationSingularNx2<float, Function2DSum1D<float, Function1DHuber<float>>>>;

// ElemOperationSingularNx2<other>
template class ProxElemOperation<float, ElemOperationSingularNx2<float, Function2DIndL1Ball<float>>>;
template class ProxElemOperation<float, ElemOperationSingularNx2<float, Function2DMoreau<float, Function2DIndL1Ball<float>>>>;

// ElemOperationEigen2x2
template class ProxElemOperation<float, ElemOperationEigen2x2<float, Function1DZero<float>>>;
template class ProxElemOperation<float, ElemOperationEigen2x2<float, Function1DAbs<float>>>;
template class ProxElemOperation<float, ElemOperationEigen2x2<float, Function1DSquare<float>>>;
template class ProxElemOperation<float, ElemOperationEigen2x2<float, Function1DIndLeq0<float>>>;
template class ProxElemOperation<float, ElemOperationEigen2x2<float, Function1DIndGeq0<float>>>;
template class ProxElemOperation<float, ElemOperationEigen2x2<float, Function1DIndEq0<float>>>;
template class ProxElemOperation<float, ElemOperationEigen2x2<float, Function1DIndBox01<float>>>;
template class ProxElemOperation<float, ElemOperationEigen2x2<float, Function1DMaxPos0<float>>>;
template class ProxElemOperation<float, ElemOperationEigen2x2<float, Function1DL0<float>>>;
template class ProxElemOperation<float, ElemOperationEigen2x2<float, Function1DHuber<float>>>;
template class ProxElemOperation<float, ElemOperationEigen2x2<float, Function1DLq<float>>>;
template class ProxElemOperation<float, ElemOperationEigen2x2<float, Function1DLqPlusEps<float>>>;
template class ProxElemOperation<float, ElemOperationEigen2x2<float, Function1DTruncQuad<float>>>;
template class ProxElemOperation<float, ElemOperationEigen2x2<float, Function1DTruncLinear<float>>>;


// ElemOperationEigen3x3
template class ProxElemOperation<float, ElemOperationEigen3x3<float, Function1DZero<float>>>;
template class ProxElemOperation<float, ElemOperationEigen3x3<float, Function1DAbs<float>>>;
template class ProxElemOperation<float, ElemOperationEigen3x3<float, Function1DSquare<float>>>;
template class ProxElemOperation<float, ElemOperationEigen3x3<float, Function1DIndLeq0<float>>>;
template class ProxElemOperation<float, ElemOperationEigen3x3<float, Function1DIndGeq0<float>>>;
template class ProxElemOperation<float, ElemOperationEigen3x3<float, Function1DIndEq0<float>>>;
template class ProxElemOperation<float, ElemOperationEigen3x3<float, Function1DIndBox01<float>>>;
template class ProxElemOperation<float, ElemOperationEigen3x3<float, Function1DMaxPos0<float>>>;
template class ProxElemOperation<float, ElemOperationEigen3x3<float, Function1DL0<float>>>;
template class ProxElemOperation<float, ElemOperationEigen3x3<float, Function1DHuber<float>>>;
template class ProxElemOperation<float, ElemOperationEigen3x3<float, Function1DLq<float>>>;
template class ProxElemOperation<float, ElemOperationEigen3x3<float, Function1DLqPlusEps<float>>>;
template class ProxElemOperation<float, ElemOperationEigen3x3<float, Function1DTruncQuad<float>>>;
template class ProxElemOperation<float, ElemOperationEigen3x3<float, Function1DTruncLinear<float>>>;


// Other
template class ProxElemOperation<float, ElemOperationIndSimplex<float>>;
template class ProxElemOperation<float, ElemOperationIndSum<float>>;
template class ProxElemOperation<float, ElemOperationMass4<float,false>>;
template class ProxElemOperation<float, ElemOperationMass5<float,false>>;
template class ProxElemOperation<float, ElemOperationMass4<float,true>>;
template class ProxElemOperation<float, ElemOperationMass5<float,true>>;

// double
// ElemOperation1D 
template class ProxElemOperation<double, ElemOperation1D<double, Function1DZero<double>>>;
template class ProxElemOperation<double, ElemOperation1D<double, Function1DAbs<double>>>;
template class ProxElemOperation<double, ElemOperation1D<double, Function1DSquare<double>>>;
template class ProxElemOperation<double, ElemOperation1D<double, Function1DIndLeq0<double>>>;
template class ProxElemOperation<double, ElemOperation1D<double, Function1DIndGeq0<double>>>;
template class ProxElemOperation<double, ElemOperation1D<double, Function1DIndEq0<double>>>;
template class ProxElemOperation<double, ElemOperation1D<double, Function1DIndBox01<double>>>;
template class ProxElemOperation<double, ElemOperation1D<double, Function1DMaxPos0<double>>>;
template class ProxElemOperation<double, ElemOperation1D<double, Function1DL0<double>>>;
template class ProxElemOperation<double, ElemOperation1D<double, Function1DHuber<double>>>;
template class ProxElemOperation<double, ElemOperation1D<double, Function1DLq<double>>>;
template class ProxElemOperation<double, ElemOperation1D<double, Function1DLqPlusEps<double>>>;
template class ProxElemOperation<double, ElemOperation1D<double, Function1DTruncQuad<double>>>;
template class ProxElemOperation<double, ElemOperation1D<double, Function1DTruncLinear<double>>>;

// ElemOperationNorm2
template class ProxElemOperation<double, ElemOperationNorm2<double, Function1DZero<double>>>;
template class ProxElemOperation<double, ElemOperationNorm2<double, Function1DAbs<double>>>;
template class ProxElemOperation<double, ElemOperationNorm2<double, Function1DSquare<double>>>;
template class ProxElemOperation<double, ElemOperationNorm2<double, Function1DIndLeq0<double>>>;
template class ProxElemOperation<double, ElemOperationNorm2<double, Function1DIndGeq0<double>>>;
template class ProxElemOperation<double, ElemOperationNorm2<double, Function1DIndEq0<double>>>;
template class ProxElemOperation<double, ElemOperationNorm2<double, Function1DIndBox01<double>>>;
template class ProxElemOperation<double, ElemOperationNorm2<double, Function1DMaxPos0<double>>>;
template class ProxElemOperation<double, ElemOperationNorm2<double, Function1DL0<double>>>;
template class ProxElemOperation<double, ElemOperationNorm2<double, Function1DHuber<double>>>;
template class ProxElemOperation<double, ElemOperationNorm2<double, Function1DLq<double>>>;
template class ProxElemOperation<double, ElemOperationNorm2<double, Function1DLqPlusEps<double>>>;
template class ProxElemOperation<double, ElemOperationNorm2<double, Function1DTruncQuad<double>>>;
template class ProxElemOperation<double, ElemOperationNorm2<double, Function1DTruncLinear<double>>>;

// ElemOperationSingularNx2<Function2DSum1D>
template class ProxElemOperation<double, ElemOperationSingularNx2<double, Function2DSum1D<double, Function1DZero<double>>>>;
template class ProxElemOperation<double, ElemOperationSingularNx2<double, Function2DSum1D<double, Function1DAbs<double>>>>;
template class ProxElemOperation<double, ElemOperationSingularNx2<double, Function2DSum1D<double, Function1DSquare<double>>>>;
template class ProxElemOperation<double, ElemOperationSingularNx2<double, Function2DSum1D<double, Function1DIndLeq0<double>>>>;
template class ProxElemOperation<double, ElemOperationSingularNx2<double, Function2DSum1D<double, Function1DIndGeq0<double>>>>;
template class ProxElemOperation<double, ElemOperationSingularNx2<double, Function2DSum1D<double, Function1DIndEq0<double>>>>;
template class ProxElemOperation<double, ElemOperationSingularNx2<double, Function2DSum1D<double, Function1DIndBox01<double>>>>;
template class ProxElemOperation<double, ElemOperationSingularNx2<double, Function2DSum1D<double, Function1DMaxPos0<double>>>>;
template class ProxElemOperation<double, ElemOperationSingularNx2<double, Function2DSum1D<double, Function1DL0<double>>>>;
template class ProxElemOperation<double, ElemOperationSingularNx2<double, Function2DSum1D<double, Function1DHuber<double>>>>;

// ElemOperationSingularNx2<other>
template class ProxElemOperation<double, ElemOperationSingularNx2<double, Function2DIndL1Ball<double>>>;
template class ProxElemOperation<double, ElemOperationSingularNx2<double, Function2DMoreau<double, Function2DIndL1Ball<double>>>>;

// ElemOperationEigen2x2
template class ProxElemOperation<double, ElemOperationEigen2x2<double, Function1DZero<double>>>;
template class ProxElemOperation<double, ElemOperationEigen2x2<double, Function1DAbs<double>>>;
template class ProxElemOperation<double, ElemOperationEigen2x2<double, Function1DSquare<double>>>;
template class ProxElemOperation<double, ElemOperationEigen2x2<double, Function1DIndLeq0<double>>>;
template class ProxElemOperation<double, ElemOperationEigen2x2<double, Function1DIndGeq0<double>>>;
template class ProxElemOperation<double, ElemOperationEigen2x2<double, Function1DIndEq0<double>>>;
template class ProxElemOperation<double, ElemOperationEigen2x2<double, Function1DIndBox01<double>>>;
template class ProxElemOperation<double, ElemOperationEigen2x2<double, Function1DMaxPos0<double>>>;
template class ProxElemOperation<double, ElemOperationEigen2x2<double, Function1DL0<double>>>;
template class ProxElemOperation<double, ElemOperationEigen2x2<double, Function1DHuber<double>>>;
template class ProxElemOperation<double, ElemOperationEigen2x2<double, Function1DLq<double>>>;
template class ProxElemOperation<double, ElemOperationEigen2x2<double, Function1DLqPlusEps<double>>>;
template class ProxElemOperation<double, ElemOperationEigen2x2<double, Function1DTruncQuad<double>>>;
template class ProxElemOperation<double, ElemOperationEigen2x2<double, Function1DTruncLinear<double>>>;


// ElemOperationEigen3x3
template class ProxElemOperation<double, ElemOperationEigen3x3<double, Function1DZero<double>>>;
template class ProxElemOperation<double, ElemOperationEigen3x3<double, Function1DAbs<double>>>;
template class ProxElemOperation<double, ElemOperationEigen3x3<double, Function1DSquare<double>>>;
template class ProxElemOperation<double, ElemOperationEigen3x3<double, Function1DIndLeq0<double>>>;
template class ProxElemOperation<double, ElemOperationEigen3x3<double, Function1DIndGeq0<double>>>;
template class ProxElemOperation<double, ElemOperationEigen3x3<double, Function1DIndEq0<double>>>;
template class ProxElemOperation<double, ElemOperationEigen3x3<double, Function1DIndBox01<double>>>;
template class ProxElemOperation<double, ElemOperationEigen3x3<double, Function1DMaxPos0<double>>>;
template class ProxElemOperation<double, ElemOperationEigen3x3<double, Function1DL0<double>>>;
template class ProxElemOperation<double, ElemOperationEigen3x3<double, Function1DHuber<double>>>;
template class ProxElemOperation<double, ElemOperationEigen3x3<double, Function1DLq<double>>>;
template class ProxElemOperation<double, ElemOperationEigen3x3<double, Function1DLqPlusEps<double>>>;
template class ProxElemOperation<double, ElemOperationEigen3x3<double, Function1DTruncQuad<double>>>;
template class ProxElemOperation<double, ElemOperationEigen3x3<double, Function1DTruncLinear<double>>>;


// other
template class ProxElemOperation<double, ElemOperationIndSimplex<double>>;
template class ProxElemOperation<double, ElemOperationIndSum<double>>;
template class ProxElemOperation<double, ElemOperationMass4<double, true>>;
template class ProxElemOperation<double, ElemOperationMass4<double, false>>;
template class ProxElemOperation<double, ElemOperationMass5<double, true>>;
template class ProxElemOperation<double, ElemOperationMass5<double, false>>;

} // namespace prost

